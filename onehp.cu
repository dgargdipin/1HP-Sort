#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include<iostream>
#include "onehp.cuh"
#include<hipcub/hipcub.hpp>
#include<algorithm>

void printArray(int* a, int n) {
	for (int i = 0; i < n; i++) {
		printf("%d ", a[i]);
	}
	printf("\n");
}

void debugArray(char a[], int* arr, int n) {
	printf("DEBUGGING %s\n", a);
	int* host_arr = new int[n];
	hipMemcpy(host_arr, arr, n * sizeof(int), hipMemcpyDeviceToHost);
	printArray(host_arr, n);
	delete[] host_arr;
}

__global__ void createHistogram(int* a, int* h, int N)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= N) return;

	int pos = a[tid];
	atomicAdd(&h[pos], 1);

}

int* one_hp_sort(int* x, int N, int minVal, int maxVal)
{
	int* y;//output
	hipMalloc(&y, sizeof(int) * N);
	int* A, * A_p;
	hipMalloc(&A, sizeof(int) * maxVal);
	hipMalloc(&A_p, sizeof(int) * maxVal);
	int numThreads = 256;
	int numBlocks = (N + numThreads - 1) / numThreads;
	createHistogram << <numBlocks, numThreads >> > (x, A, N);
	prefix_sum_on_gpu(A, A_p, maxVal);
	OneHpTail << < (maxVal - minVal + 255) / 256, 256 >> > (minVal, maxVal, A_p, y);
	hipFree(A);
	hipFree(A_p);
	return y;


}

void prefix_sum_on_gpu(int* data, int* output, int size) {
	void* d_temp_storage = NULL;
	size_t   temp_storage_bytes = 0;
	hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, data, output, size);
	// Allocate temporary storage for inclusive prefix sum
	hipMalloc(&d_temp_storage, temp_storage_bytes);
	// Run inclusive prefix sum
	hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, data, output, size);
	//printf("Successfully prefixed sum");
}


__global__ void OneHpTail(int minVal, int maxVal, int* Ap, int* y) {
	int i = minVal + blockIdx.x * blockDim.x + threadIdx.x;
	if (i > maxVal)return;
	if (i == minVal) {
		if (Ap[minVal])y[0] = minVal;
	}
	else if (Ap[i - 1] != Ap[i])y[Ap[i - 1]] = i;
}

void test_one_hp(int* d_x, int N, int minVal, int maxVal, bool verbose) {
	printf("-------------Testing 1HP algorithm---------------------\n");
	test_sort(d_x, N, minVal, maxVal, &one_hp_sort, verbose);
};
void test_cubsort(int* d_x, int N, int minVal, int maxVal, bool verbose)
{
	printf("-------------Testing CubSort algorithm---------------------\n");
	test_sort(d_x, N, minVal, maxVal, &cubsort, verbose);
}


int* cubsort(int* d_x, int N, int minVal, int maxVal)
{
	// Declare, allocate, and initialize device-accessible pointers for sorting data
	int  num_items = N;          // e.g., 7
	int* d_keys_in = d_x;         // e.g., [8, 6, 7, 5, 3, 0, 9]
	int* d_keys_out;        // e.g., [        ...        ]
	hipMalloc(&d_keys_out, sizeof(int) * N);
	// Determine temporary device storage requirements
	void* d_temp_storage = NULL;
	size_t   temp_storage_bytes = 0;
	hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, num_items);
	// Allocate temporary storage
	hipMalloc(&d_temp_storage, temp_storage_bytes);
	// Run sorting operation
	hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, num_items);
	// d_keys_out            <-- [0, 3, 5, 6, 7, 8, 9]
	return d_keys_out;
}




void test_sort(int* d_x, int N, int minVal, int maxVal, int* (*func)(int*, int, int, int), bool verbose) {
	float milliseconds = 0;
	char input_str[] = "input:";
	if (verbose)debugArray(input_str, d_x, N);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	int* d_y = func(d_x, N, minVal, maxVal);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);

	char output_str[] = "output:";
	if (verbose)debugArray(output_str, d_y, N);
	printf("Took %f milliseconds\n", milliseconds);
	hipFree(d_y);
}

std::vector<int> generate_random_unique_array(int N, int M) {
	std::vector<int> x(M);


	for (int i = 0; i < M; i++) {
		x[i] = i;
	}
	std::srand(unsigned(std::time(0)));
	std::random_shuffle(x.begin(), x.end());
	x = std::vector<int>(x.begin(), x.begin() + N);
	return x;

};
void getInput(int& N, int& M, bool& verbose) {
	std::cout << "Enter the length of random array: ";
	std::cin >> N;
	std::cout << std::endl;
	std::cout << "Enter the range of random array: ";
	std::cin >> M;
	std::cout << std::endl;
	std::cout << "Do you want to verbose output? (y/n): ";
	char verboseChar;
	std::cin >> verboseChar;
	verbose = (verboseChar == 'y');
}